#include "hip/hip_runtime.h"
#include "conv.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#define ERROR_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::exit(EXIT_FAILURE); \
    }


void launch_kernel(
    const int8_t* in,
    int8_t* out,
    const conv2d& conv
) {
    // sizes
    int in_size = conv.C_in * conv.H_in * conv.W_in * conv.batch_size;
    int out_size = conv.C_out * conv.H_out * conv.W_out * conv.batch_size;
    int weights_size = conv.C_out * conv.C_in * conv.k_h * conv.k_w;
    int bias_size = conv.C_out;

    // GPU memory allocation
    int8_t* kernel_in, *kernel_out, *kernel_weights;
    float* kernel_bias;
    conv2d* kernel_conv;

    ERROR_CHECK(hipMalloc(&kernel_in, in_size * sizeof(int8_t)));
    ERROR_CHECK(hipMalloc(&kernel_out, out_size * sizeof(int8_t)));
    ERROR_CHECK(hipMalloc(&kernel_weights, weights_size * sizeof(int8_t)));
    ERROR_CHECK(hipMalloc(&kernel_bias, bias_size * sizeof(float)));
    ERROR_CHECK(hipMalloc(&kernel_conv, sizeof(conv2d)));

    // Get current CUDA stream
    hipStream_t curr_stream = at::cuda::getCurrentCUDAStream();

    // Copy input, weights, and bias to GPU
    ERROR_CHECK(hipMemcpyAsync(kernel_in, in, in_size * sizeof(int8_t), hipMemcpyHostToDevice, curr_stream));
    ERROR_CHECK(hipMemcpyAsync(kernel_weights, conv.weights, weights_size * sizeof(int8_t), hipMemcpyHostToDevice, curr_stream));
    ERROR_CHECK(hipMemcpyAsync(kernel_bias, conv.bias, bias_size * sizeof(float), hipMemcpyHostToDevice, curr_stream));

    // Copy conv to GPU
    conv2d conv_copy = conv;
    conv_copy.weights = kernel_weights;
    conv_copy.bias = kernel_bias;
    ERROR_CHECK(hipMemcpyAsync(kernel_conv, &conv_copy, sizeof(conv2d), hipMemcpyHostToDevice, curr_stream));

    // Launch kernel
    int block_dim = 256; // Number of threads per block
    dim3 blockDim(block_dim);
    dim3 gridDim((conv.H_out * conv.W_out + block_dim - 1) / block_dim, conv.C_out, conv.batch_size); // equivalent to ceil(H_out * W_out / block_dim)
    kernel<<<gridDim, blockDim, 0, curr_stream>>>(kernel_in, kernel_out, *kernel_conv);

    // Copy output
    ERROR_CHECK(hipMemcpyAsync(out, kernel_out, out_size * sizeof(int8_t), hipMemcpyDeviceToHost, curr_stream));

    // Free GPU memory
    ERROR_CHECK(hipFree(kernel_in));
    ERROR_CHECK(hipFree(kernel_out));
    ERROR_CHECK(hipFree(kernel_weights));
    ERROR_CHECK(hipFree(kernel_bias));
    ERROR_CHECK(hipFree(kernel_conv));

    // Synchronize
    ERROR_CHECK(hipStreamSynchronize(curr_stream));
}